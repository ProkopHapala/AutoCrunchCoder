#include "hip/hip_runtime.h"
// nbody_coulomb.cu

#define COULOMB_CONST 14.3996448915
#define BLOCK_SIZE 32

__global__ void nbody_coulomb(
    int n, 
    const float4* pos, 
    const float4* params, 
    float4* fe_out
) {
    int gid = blockIdx.x * blockDim.x + threadIdx.x;

    if (gid >= n) return;  // Early exit if gid is out of bounds

    extern __shared__ float4 shared_mem[];
    float4* local_pos = (float4*)shared_mem;
    float4* local_params = (float4*)&shared_mem[blockDim.x];

    float3 pi   = make_float3(pos[gid].x, pos[gid].y, pos[gid].z);
    float4 pari = params[gid];
    float4 fe   = make_float4(0.0f, 0.0f, 0.0f, 0.0f);

    // Loop over all blocks
    for (int wg = 0; wg < n; wg += blockDim.x) {
        int local_idx = wg + threadIdx.x;

        // Load data into shared memory if within bounds
        if (local_idx < n) {
            local_pos[threadIdx.x] = pos[local_idx];
            local_params[threadIdx.x] = params[local_idx];
        }

        // Synchronize to ensure all shared memory is populated
        __syncthreads();

        // Compute interactions with particles in the shared memory
        for (int j = 0; j < blockDim.x; j++) {
            int global_j = wg + j;

            if (global_j >= n || gid == global_j) continue;  // Skip out-of-bounds and self-interaction

            float3 pj   = make_float3(local_pos[j].x, local_pos[j].y, local_pos[j].z);
            float4 parj = local_params[j];

            float3 d  = make_float3(pj.x - pi.x, pj.y - pi.y, pj.z - pi.z);
            float ir2 = dot(d, d) + 1e-32f;
            float ir  = sqrtf(ir2);
            float qq  = pari.z * parj.z;
            float E   = COULOMB_CONST * qq * ir;

            fe.x += d.x * (E * ir2);
            fe.y += d.y * (E * ir2);
            fe.z += d.z * (E * ir2);
            fe.w += E;
        }

        // Synchronize before the next iteration
        __syncthreads();
    }

    // Write the computed force to global memory
    fe_out[gid] = fe;
}
